#include "hip/hip_runtime.h"
#include "calc.cuh"
#include "launchable.cuh"
#include "cudaerrchk.cuh"

Calc::Calc(LaunchConfig lc,
	InputFile inf,
	double* IGmax,
	void (*progressCallback)(int, int, int, int)) :
	lc(lc), inf(inf), IGmax(IGmax), pC(progressCallback),
	scheduler(lc.dim, (inf.vars + lc.tileSize - 1) / lc.tileSize, pC),
	discretizer(lc, inf, &scheduler) {

	df = discretizer.getDataFile();

	memset(IGmax, 0, inf.vars * sizeof(double));

	int workerCount;
	CUDA(hipGetDeviceCount(&workerCount));

	for (int i = 0; i < workerCount; ++i) {
		workerlist.push_back(new Worker(inf.vars, i, lc, this, &scheduler, df));
	}

	if (workerlist.size() == 0) {
		return;
	}

	threads.push_back(std::thread(Launch, &discretizer));

	for (auto i : this->workerlist) {
		threads.push_back(std::thread(Launch, i));
	}
	for (auto& t : this->threads) {
		t.join();
	}

	for (auto i : this->workerlist) {
		delete i;
	}
}

void Calc::returnResults(float* IG) {
	mutex.lock();
	for (int i = 0; i < inf.vars; i++) {
		IGmax[i] = std::max(IGmax[i], (double) IG[i]);
	}
	mutex.unlock();
}

