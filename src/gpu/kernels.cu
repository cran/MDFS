#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "kernels2D.cuh"
#include "kernels3D.cuh"
#include "kernels4D.cuh"
#include "kernels5D.cuh"
#include "tableskernel.cuh"
#include "splitkernel.cuh"

std::vector<std::pair<KernelParam, void(*)(KernelParam, hipStream_t)>> kernels;

std::ostream& operator<< (std::ostream& out, KernelParam const& prop) {
	out << "tileSize:" << prop.tileSize << " ";
	out << "dim:" << prop.dim << " ";
	out << "div:" << prop.div << " ";
	out << "rm:" << prop.rm << " ";
	out << "bf:" << prop.bf << " ";
	out << "tablesKernel:" << prop.tablesKernel << " ";
	out << (prop.objs[0] < (1 << 16) ? "objc0 < 2^16 " : "objc0 >= 2^16 ");
	out << (prop.objs[1] < (1 << 16) ? "objc1 < 2^16 " : "objc1 >= 2^16 ");
	return out;
}

bool operator==(const KernelParam& lhs, const KernelParam& rhs) {
	bool lobjs = (lhs.objs[0] < (1 << 16)) && (lhs.objs[1] < (1 << 16));
	bool robjs = (rhs.objs[0] < (1 << 16)) && (rhs.objs[1] < (1 << 16));

	return (lobjs == robjs) &&
		(lhs.tileSize == rhs.tileSize) &&
		(lhs.dim == rhs.dim) &&
		(lhs.div == rhs.div) &&
		(lhs.rm == rhs.rm) &&
		(lhs.bf == rhs.bf) &&
		(lhs.tablesKernel == rhs.tablesKernel);
}

KernelParam::KernelParam(LaunchConfig lc, bool tablesKernel, int index,
	int vars, uint64_t** data, uint64_t** counters,
	std::vector<int> offset, int packs0, int packs1,
	int objs0, int objs1, float* IG)
	: tileSize(lc.tileSize), dim(lc.dim), div(lc.div),
	rm(lc.rm), bf(lc.bf), tablesKernel(tablesKernel),
	index(index), disc(lc.disc), vars(vars), IG(IG) {

	objs[0] = objs0;
	objs[1] = objs1;

	for (int i = 0; i < dim; i++) {
		this->data[i] = data[i];
		this->counters[i] = counters[i];
		this->offset[i] = offset[i];
	}

	packs[0] = packs0;
	packs[1] = packs1;

	const float objsmin = std::min(objs0, objs1);

	for (int i = 0; i < 2; i++) {
		pseudo[i] = ((float)objs[i] / objsmin) * lc.pseudo;
	}
}

KernelParam::KernelParam(int tileSize, int dim, int div, ReduceMethod rm,
	BinaryFormat bf, bool tablesKernel, int objs0)
	: tileSize(tileSize), dim(dim), div(div), rm(rm), bf(bf),
	tablesKernel(tablesKernel)  {

	for (int i = 0; i < 2; i++) {
		objs[i] = objs0;
	}
}

bool init() {
#define KERNELS2D(DIV) \
	kernels.push_back(std::make_pair(KernelParam(512, 2, (DIV), RM_AVG, BF_SHIFT, false,     0), kernel2DWrapper<512, (DIV), 1, 1>));\
	kernels.push_back(std::make_pair(KernelParam(512, 2, (DIV), RM_MAX, BF_SHIFT, false,     0), kernel2DWrapper<512, (DIV), 0, 1>));\
	kernels.push_back(std::make_pair(KernelParam(512, 2, (DIV), RM_AVG, BF_SHIFT, false, 1<<16), kernel2DWrapper<512, (DIV), 1, 0>));\
	kernels.push_back(std::make_pair(KernelParam(512, 2, (DIV), RM_MAX, BF_SHIFT, false, 1<<16), kernel2DWrapper<512, (DIV), 0, 0>));
#define KERNELS3D(DIV) \
	kernels.push_back(std::make_pair(KernelParam(64, 3, (DIV), RM_AVG, BF_SHIFT, false,     0), kernel3DWrapper<64, (DIV), 1, 1>));\
	kernels.push_back(std::make_pair(KernelParam(64, 3, (DIV), RM_MAX, BF_SHIFT, false,     0), kernel3DWrapper<64, (DIV), 0, 1>));\
	kernels.push_back(std::make_pair(KernelParam(64, 3, (DIV), RM_AVG, BF_SHIFT, false, 1<<16), kernel3DWrapper<64, (DIV), 1, 0>));\
	kernels.push_back(std::make_pair(KernelParam(64, 3, (DIV), RM_MAX, BF_SHIFT, false, 1<<16), kernel3DWrapper<64, (DIV), 0, 0>));

#define SPLITKERNELS(TS, DIM, DIV, BITS) \
	kernels.push_back(std::make_pair(KernelParam((TS),(DIM),(DIV),RM_AVG,BF_SPLIT,false,    0),splitKernelWrapper<(TS),(DIM),(DIV),(BITS),1,1>));\
	kernels.push_back(std::make_pair(KernelParam((TS),(DIM),(DIV),RM_MAX,BF_SPLIT,false,    0),splitKernelWrapper<(TS),(DIM),(DIV),(BITS),0,1>));\
	kernels.push_back(std::make_pair(KernelParam((TS),(DIM),(DIV),RM_AVG,BF_SPLIT,false,1<<16),splitKernelWrapper<(TS),(DIM),(DIV),(BITS),1,0>));\
	kernels.push_back(std::make_pair(KernelParam((TS),(DIM),(DIV),RM_MAX,BF_SPLIT,false,1<<16),splitKernelWrapper<(TS),(DIM),(DIV),(BITS),0,0>));

	KERNELS2D(10)
	KERNELS2D(11)
	KERNELS2D(12)
	KERNELS2D(13)
	KERNELS2D(14)
	KERNELS2D(15)

	KERNELS3D(5)

	SPLITKERNELS(512, 2, 1, 1)
	SPLITKERNELS(512, 2, 2, 2)
	SPLITKERNELS(512, 2, 3, 2)
	SPLITKERNELS(512, 2, 4, 4)
	SPLITKERNELS(512, 2, 5, 4)
	SPLITKERNELS(512, 2, 6, 4)
	SPLITKERNELS(512, 2, 7, 4)
	SPLITKERNELS(512, 2, 8, 4)
	SPLITKERNELS(512, 2, 9, 4)

	SPLITKERNELS(64, 3, 1, 1)
	SPLITKERNELS(64, 3, 2, 2)
	SPLITKERNELS(64, 3, 3, 2)
	SPLITKERNELS(64, 3, 4, 4)

	SPLITKERNELS(32, 4, 1, 1)
	SPLITKERNELS(32, 4, 2, 2)
	SPLITKERNELS(32, 4, 3, 2)

	SPLITKERNELS(8, 5, 1, 1)
	SPLITKERNELS(8, 5, 2, 2)

	// FIXME?
	kernels.push_back(std::make_pair(KernelParam(64, 3, 1, RM_AVG, BF_SPLIT, true,     0), tablesKernelWrapper<64, 2, 1, 1, 1>));
	kernels.push_back(std::make_pair(KernelParam(64, 3, 1, RM_MAX, BF_SPLIT, true,     0), tablesKernelWrapper<64, 2, 1, 1, 1>));
	kernels.push_back(std::make_pair(KernelParam(64, 3, 1, RM_AVG, BF_SPLIT, true, 1<<16), tablesKernelWrapper<64, 2, 1, 1, 0>));
	kernels.push_back(std::make_pair(KernelParam(64, 3, 1, RM_MAX, BF_SPLIT, true, 1<<16), tablesKernelWrapper<64, 2, 1, 1, 0>));

	kernels.push_back(std::make_pair(KernelParam(64, 3, 2, RM_AVG, BF_SPLIT, true,     0), tablesKernelWrapper<64, 2, 2, 2, 1>));
	kernels.push_back(std::make_pair(KernelParam(64, 3, 2, RM_MAX, BF_SPLIT, true,     0), tablesKernelWrapper<64, 2, 2, 2, 1>));
	kernels.push_back(std::make_pair(KernelParam(64, 3, 2, RM_AVG, BF_SPLIT, true, 1<<16), tablesKernelWrapper<64, 2, 2, 2, 0>));
	kernels.push_back(std::make_pair(KernelParam(64, 3, 2, RM_MAX, BF_SPLIT, true, 1<<16), tablesKernelWrapper<64, 2, 2, 2, 0>));

	kernels.push_back(std::make_pair(KernelParam(64, 3, 3, RM_AVG, BF_SPLIT, true,     0), tablesKernelWrapper<64, 2, 3, 2, 1>));
	kernels.push_back(std::make_pair(KernelParam(64, 3, 3, RM_MAX, BF_SPLIT, true,     0), tablesKernelWrapper<64, 2, 3, 2, 1>));
	kernels.push_back(std::make_pair(KernelParam(64, 3, 3, RM_AVG, BF_SPLIT, true, 1<<16), tablesKernelWrapper<64, 2, 3, 2, 0>));
	kernels.push_back(std::make_pair(KernelParam(64, 3, 3, RM_MAX, BF_SPLIT, true, 1<<16), tablesKernelWrapper<64, 2, 3, 2, 0>));

	kernels.push_back(std::make_pair(KernelParam(64, 3, 4, RM_AVG, BF_SPLIT, true,     0), tablesKernelWrapper<64, 2, 4, 4, 1>));
	kernels.push_back(std::make_pair(KernelParam(64, 3, 4, RM_MAX, BF_SPLIT, true,     0), tablesKernelWrapper<64, 2, 4, 4, 1>));
	kernels.push_back(std::make_pair(KernelParam(64, 3, 4, RM_AVG, BF_SPLIT, true, 1<<16), tablesKernelWrapper<64, 2, 4, 4, 0>));
	kernels.push_back(std::make_pair(KernelParam(64, 3, 4, RM_MAX, BF_SPLIT, true, 1<<16), tablesKernelWrapper<64, 2, 4, 4, 0>));

	kernels.push_back(std::make_pair(KernelParam(32, 4, 1, RM_AVG, BF_SPLIT, true,     0), tablesKernelWrapper<32, 3, 1, 1, 1>));
	kernels.push_back(std::make_pair(KernelParam(32, 4, 1, RM_MAX, BF_SPLIT, true,     0), tablesKernelWrapper<32, 3, 1, 1, 1>));
	kernels.push_back(std::make_pair(KernelParam(32, 4, 1, RM_AVG, BF_SPLIT, true, 1<<16), tablesKernelWrapper<32, 3, 1, 1, 0>));
	kernels.push_back(std::make_pair(KernelParam(32, 4, 1, RM_MAX, BF_SPLIT, true, 1<<16), tablesKernelWrapper<32, 3, 1, 1, 0>));

	kernels.push_back(std::make_pair(KernelParam(32, 4, 2, RM_AVG, BF_SPLIT, true,     0), tablesKernelWrapper<32, 3, 2, 2, 1>));
	kernels.push_back(std::make_pair(KernelParam(32, 4, 2, RM_MAX, BF_SPLIT, true,     0), tablesKernelWrapper<32, 3, 2, 2, 1>));
	kernels.push_back(std::make_pair(KernelParam(32, 4, 2, RM_AVG, BF_SPLIT, true, 1<<16), tablesKernelWrapper<32, 3, 2, 2, 0>));
	kernels.push_back(std::make_pair(KernelParam(32, 4, 2, RM_MAX, BF_SPLIT, true, 1<<16), tablesKernelWrapper<32, 3, 2, 2, 0>));

	kernels.push_back(std::make_pair(KernelParam(32, 4, 3, RM_AVG, BF_SPLIT, true,     0), tablesKernelWrapper<32, 3, 3, 2, 1>));
	kernels.push_back(std::make_pair(KernelParam(32, 4, 3, RM_MAX, BF_SPLIT, true,     0), tablesKernelWrapper<32, 3, 3, 2, 1>));
	kernels.push_back(std::make_pair(KernelParam(32, 4, 3, RM_AVG, BF_SPLIT, true, 1<<16), tablesKernelWrapper<32, 3, 3, 2, 0>));
	kernels.push_back(std::make_pair(KernelParam(32, 4, 3, RM_MAX, BF_SPLIT, true, 1<<16), tablesKernelWrapper<32, 3, 3, 2, 0>));

	kernels.push_back(std::make_pair(KernelParam(8, 5, 1, RM_AVG, BF_SPLIT, true,     0), tablesKernelWrapper<8, 4, 1, 1, 1>));
	kernels.push_back(std::make_pair(KernelParam(8, 5, 1, RM_MAX, BF_SPLIT, true,     0), tablesKernelWrapper<8, 4, 1, 1, 1>));
	kernels.push_back(std::make_pair(KernelParam(8, 5, 1, RM_AVG, BF_SPLIT, true, 1<<16), tablesKernelWrapper<8, 4, 1, 1, 0>));
	kernels.push_back(std::make_pair(KernelParam(8, 5, 1, RM_MAX, BF_SPLIT, true, 1<<16), tablesKernelWrapper<8, 4, 1, 1, 0>));

	kernels.push_back(std::make_pair(KernelParam(8, 5, 2, RM_AVG, BF_SPLIT, true,     0), tablesKernelWrapper<8, 4, 2, 2, 1>));
	kernels.push_back(std::make_pair(KernelParam(8, 5, 2, RM_MAX, BF_SPLIT, true,     0), tablesKernelWrapper<8, 4, 2, 2, 1>));
	kernels.push_back(std::make_pair(KernelParam(8, 5, 2, RM_AVG, BF_SPLIT, true, 1<<16), tablesKernelWrapper<8, 4, 2, 2, 0>));
	kernels.push_back(std::make_pair(KernelParam(8, 5, 2, RM_MAX, BF_SPLIT, true, 1<<16), tablesKernelWrapper<8, 4, 2, 2, 0>));

	return true;
}

static const bool doneInit = init();

