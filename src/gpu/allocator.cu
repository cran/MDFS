#include <iostream>
#include <cstdio>
#include "cudaerrchk.cuh"
#include "allocator.cuh"

Allocator _alloc;

static std::string ptrInfo(std::string name,
	void* ptr,
	std::size_t size,
	int line,
	std::string file) {
	return name + ":" + std::to_string((long long) ptr)
		+ ", size:" + std::to_string(size)
		+ ", line:" + std::to_string(line)
		+ ", file:" + file;
}

void* Allocator::mallocHost(std::size_t size, int line, std::string file) {
	void* ptr = malloc(size);

	//std::cout << "@1@" << ptr << " " << size << " " << file << line << "\n\n";

	loc[ptr] = ptrInfo("host", ptr, size, line, file);
	return ptr;
}

void* Allocator::mallocPinned(std::size_t size, int line, std::string file) {
	void* ptr;
	CUDA(hipHostMalloc(&ptr, size));

	//std::cout << "@2@" << ptr << "\n\n";

	loc[ptr] = ptrInfo("pinned", ptr, size, line, file);
	return ptr;
}

void* Allocator::mallocDevice(std::size_t size, int line, std::string file) {
	void* ptr;
	CUDA(hipMalloc(&ptr, size));
	loc[ptr] = ptrInfo("device", ptr, size, line, file);
	return ptr;
}

void Allocator::freeHost(void* ptr) {
	free(ptr);
	loc.erase(ptr);
}

void Allocator::freePinned(void* ptr) {
	//printf("freePinned: %lld\n", ptr);
	CUDA(hipHostFree(ptr));
	loc.erase(ptr);
}
void Allocator::freeDevice(void* ptr) {
	CUDA(hipFree(ptr));
	loc.erase(ptr);
}

Allocator::~Allocator() {
	if (!loc.empty()) {
		for (auto it = loc.begin(); it != loc.end(); ++it) {
			//std::cerr << (*it).second << std::endl;
		}
	}
}
